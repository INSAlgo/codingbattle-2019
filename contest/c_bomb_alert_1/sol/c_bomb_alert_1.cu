
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <vector>

/* CUDA version of the solution for exercise C.
 * Actually doesn't provide interesting speedups with the given input/output
 * samples because:
 * A- the samples have very small problem dimensions so the overhead of the
 *    first CUDA API call is by far longer than the actual execution
 * B- more time is spent reading the inputs than doing actual computations
 */

/* Helper function to calculate the Euclidean distance between two points
 * on a 2D plane */
inline __device__ float euclidean_distance(float x1, float y1, float x2,
                                           float y2) {
  float xd = x1 - x2, yd = y1 - y2;
  return sqrt(xd * xd + yd * yd);
}

__global__ void suspects_kernel(const float *x, const float *y, bool *results,
                                int n_sus, int n_pos, float xb, float yb) {
  int pos_idx = blockIdx.x * (blockDim.x - 1) + threadIdx.x;
  int sus_idx = blockIdx.y * blockDim.y + threadIdx.y;

  // Load block in shared memory
  __shared__ float x_shared[256];
  __shared__ float y_shared[256];
  {
    int shared_load_idx = threadIdx.y * blockDim.x + threadIdx.x;
    int global_load_idx = sus_idx * n_pos + pos_idx;
    bool do_load = sus_idx < n_sus && pos_idx < n_pos;
    x_shared[shared_load_idx] = do_load ? x[global_load_idx] : 0.0;
    y_shared[shared_load_idx] = do_load ? y[global_load_idx] : 0.0;
  }
  __syncthreads();

  /* Compute the sum of the distances from the position to bomb and the bomb
   * to the next position */
  float dist = 101.0;
  if (sus_idx < n_sus && pos_idx < n_pos - 1 && threadIdx.x < blockDim.x - 1) {
    int shared_idx = threadIdx.y * blockDim.x + threadIdx.x;
    dist =
        euclidean_distance(x_shared[shared_idx], y_shared[shared_idx], xb, yb) +
        euclidean_distance(x_shared[shared_idx + 1], y_shared[shared_idx + 1],
                           xb, yb);
  }
  if (dist <= 100.0)
    results[sus_idx] = true;
}

int main(int argc, char *argv[]) {
  int n_sus, n_pos;
  float xb, yb;

  std::cin >> xb >> yb >> n_sus >> n_pos;

  // Allocate device memory
  float *x_d, *y_d;
  bool *res_d;
  hipMalloc((void **)&x_d, n_sus * n_pos * sizeof(float));
  hipMalloc((void **)&y_d, n_sus * n_pos * sizeof(float));
  hipMalloc((void **)&res_d, n_sus * sizeof(bool));

  // Parse input
  std::vector<float> x_h = std::vector<float>(n_sus * n_pos);
  std::vector<float> y_h = std::vector<float>(n_sus * n_pos);
  std::vector<std::string> names = std::vector<std::string>(n_sus);
  for (int i = 0; i < n_sus; i++) {
    std::cin >> names[i];
    for (int j = i * n_pos; j < (i + 1) * n_pos; j++) {
      std::cin >> x_h[j] >> y_h[j];
    }
  }

  // Transfer data to the device
  hipMemcpy(x_d, x_h.data(), n_sus * n_pos * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(y_d, y_h.data(), n_sus * n_pos * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemset(res_d, 0, n_sus * sizeof(bool));

  /* Launch kernel to compute suspects
   * Note: grid and block dims should be fine-tuned for the problem dimensions
   */
  dim3 block(16, 16, 1);
  dim3 grid((n_pos - 1) / (block.x - 1) + 1, (n_sus - 1) / block.y + 1, 1);
  suspects_kernel<<<grid, block>>>(x_d, y_d, res_d, n_sus, n_pos, xb, yb);

  // Copy results to host
  bool res_h[n_sus];
  hipMemcpy(res_h, res_d, n_sus * sizeof(bool), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(x_d);
  hipFree(y_d);
  hipFree(res_d);

  // Output results
  for (int i = 0; i < n_sus; i++) {
    if (res_h[i])
      std::cout << names[i] << (i < n_sus ? " " : "");
  }
  std::cout << std::endl;

  return 0;
}
